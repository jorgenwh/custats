#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"
#include "functions.h"

void poisson_logpmf_hh2h(const int *k, const float *r, float *out, const int size)
{
  int *k_d;
  float *r_d, *out_d;
  cuda_errchk(hipMalloc(&k_d, size*sizeof(int)));
  cuda_errchk(hipMalloc(&r_d, size*sizeof(float)));
  cuda_errchk(hipMalloc(&out_d, size*sizeof(float)));
  cuda_errchk(hipMemcpy(k_d, k, size*sizeof(int), hipMemcpyHostToDevice));
  cuda_errchk(hipMemcpy(r_d, r, size*sizeof(float), hipMemcpyHostToDevice));

  kernels::call_poisson_logpmf_kernel(k_d, r_d, out_d, size);

  cuda_errchk(hipMemcpy(out, out_d, size*sizeof(float), hipMemcpyDeviceToHost));
}

void poisson_logpmf_dh2d(const int *k, const float *r, float *out, const int size)
{
  float *r_d;
  cuda_errchk(hipMalloc(&r_d, size*sizeof(float)));
  cuda_errchk(hipMemcpy(r_d, r, size*sizeof(float), hipMemcpyHostToDevice));

  kernels::call_poisson_logpmf_kernel(k, r_d, out, size);
}

void poisson_logpmf_hd2d(const int *k, const float *r, float *out, const int size)
{
  int *k_d;
  cuda_errchk(hipMalloc(&k_d, size*sizeof(int)));
  cuda_errchk(hipMemcpy(k_d, k, size*sizeof(int), hipMemcpyHostToDevice));

  kernels::call_poisson_logpmf_kernel(k_d, r, out, size);
}

void poisson_logpmf_dd2d(const int *k, const float *r, float *out, const int size)
{
  kernels::call_poisson_logpmf_kernel(k, r, out, size);
}
