#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"
#include "functions.h"

void poisson_logpmf_hh2h(const int *k, const double *r, double *out, const int size)
{
  int *k_d;
  double *r_d, *out_d;
  cuda_errchk(hipMalloc(&k_d, size*sizeof(int)));
  cuda_errchk(hipMalloc(&r_d, size*sizeof(double)));
  cuda_errchk(hipMalloc(&out_d, size*sizeof(double)));
  cuda_errchk(hipMemcpy(k_d, k, size*sizeof(int), hipMemcpyHostToDevice));
  cuda_errchk(hipMemcpy(r_d, r, size*sizeof(double), hipMemcpyHostToDevice));

  kernels::call_poisson_logpmf_kernel(k_d, r_d, out_d, size);

  cuda_errchk(hipMemcpy(out, out_d, size*sizeof(double), hipMemcpyDeviceToHost));
  cuda_errchk(hipFree(k_d));
  cuda_errchk(hipFree(r_d));
  cuda_errchk(hipFree(out_d));
}

void poisson_logpmf_dh2d(const int *k, const double *r, double *out, const int size)
{
  double *r_d;
  cuda_errchk(hipMalloc(&r_d, size*sizeof(double)));
  cuda_errchk(hipMemcpy(r_d, r, size*sizeof(double), hipMemcpyHostToDevice));

  kernels::call_poisson_logpmf_kernel(k, r_d, out, size);

  cuda_errchk(hipFree(r_d));
}

void poisson_logpmf_hd2d(const int *k, const double *r, double *out, const int size)
{
  int *k_d;
  cuda_errchk(hipMalloc(&k_d, size*sizeof(int)));
  cuda_errchk(hipMemcpy(k_d, k, size*sizeof(int), hipMemcpyHostToDevice));

  kernels::call_poisson_logpmf_kernel(k_d, r, out, size);

  cuda_errchk(hipFree(k_d));
}

void poisson_logpmf_dd2d(const int *k, const double *r, double *out, const int size)
{
  kernels::call_poisson_logpmf_kernel(k, r, out, size);
}

void poisson_logpmf_experimental(
    unsigned int *observed_counts, float *counts, unsigned int n_counts,
    float base_lambda, float error_rate, float *out)
{
  kernels::call_poisson_logpmf_experimental_kernel(
      observed_counts, counts, n_counts, base_lambda, error_rate, out);
}

