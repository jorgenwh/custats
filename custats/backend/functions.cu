#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"
#include "functions.h"

void poisson_logpmf_hh2h(const int *k, const double *r, double *out, const int size)
{
  int *k_d;
  double *r_d, *out_d;
  hipMalloc(&k_d, size*sizeof(int));
  hipMalloc(&r_d, size*sizeof(double));
  hipMalloc(&out_d, size*sizeof(double));
  hipMemcpy(k_d, k, size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(r_d, r, size*sizeof(double), hipMemcpyHostToDevice);

  kernels::call_poisson_logpmf_kernel(k_d, r_d, out_d, size);

  hipMemcpy(out, out_d, size*sizeof(double), hipMemcpyDeviceToHost);
}

void poisson_logpmf_dh2d(const int *k, const double *r, double *out, const int size)
{
  double *r_d;
  hipMalloc(&r_d, size*sizeof(double));
  hipMemcpy(r_d, r, size*sizeof(double), hipMemcpyHostToDevice);

  kernels::call_poisson_logpmf_kernel(k, r_d, out, size);
}

void poisson_logpmf_hd2d(const int *k, const double *r, double *out, const int size)
{
  int *k_d;
  hipMalloc(&k_d, size*sizeof(int));
  hipMemcpy(k_d, k, size*sizeof(int), hipMemcpyHostToDevice);

  kernels::call_poisson_logpmf_kernel(k_d, r, out, size);
}

void poisson_logpmf_dd2d(const int *k, const double *r, double *out, const int size)
{
  kernels::call_poisson_logpmf_kernel(k, r, out, size);
}
